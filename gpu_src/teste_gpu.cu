#include<stdio.h>
#include<iostream>
#include<string.h>
#include "aes_gpu.h"

using namespace std;

#define MAX_BUFFER_SIZE  536870912
#define MAX_THR_PBLK 1024


__host__
void printState(unsigned char * state, int size) {
  for(int i = 0; i < size; i++)
    printf("%X ", state[i]);
  printf("\n");
}

int main(int argc, char ** argv) {
  long buffSize = 0L, //Tamanho do arquivo
       bytesRead = 0L,
       fileSize = 0L; //Quantidade de bytes lidos pelo fread

  FILE * fin; // Pointer para o arquivo
  unsigned char * buffer = NULL; //Bytes do arquivo
  unsigned char * buffGPU = NULL;
  unsigned char * keysGPU = NULL;

  unsigned char key[16] = {
    1, 2, 3, 4,
    5, 6, 7, 8,
    9, 10, 11, 12,
    13, 14, 15, 16
  };

  fin = fopen(argv[1], "r"); //Abre o arquivo como leitura

  //Verificar se o arquivo foi 'aberto'
  if (fin == NULL) {
    cout << "Ocorreu uma falha ao tentar abrir o arquivo " << argv[1] << endl;
    return -1;
  }

  /**
    Descobre o tamanho do arquivo para debug
  */
  fseek(fin, 0L, SEEK_END);
  fileSize = ftell(fin) - 1;
  rewind(fin);

  //Expansão de chaves
  unsigned char exp_key[EXP_KEY_SIZE];
  addKeyExpansion(key, exp_key);

  //Lê os bytes do arquivo e adiciona padding caso necessário
  buffer = new unsigned char[MAX_BUFFER_SIZE];
  bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
  hipMalloc((void**)&buffGPU, sizeof(unsigned char) * MAX_BUFFER_SIZE);
  hipMalloc((void**)&keysGPU, sizeof(unsigned char) * EXP_KEY_SIZE);

  while (bytesRead > 0) {
    if (bytesRead < MAX_BUFFER_SIZE)
      bytesRead -= 1; /* Se ler o último bloco do arquivo, desconsidera o EOF */

    buffSize = bytesRead;

    //Verifica se o tamanho do arquivo é múltiplo de 16. Caso não for, encontra o próximo múltiplo de 16 a partir
    //do tamanho do arquivo
    if (buffSize % 16 != 0)
      buffSize = buffSize + (16 - (buffSize % 16));

    if (bytesRead < buffSize)
      memset((buffer + bytesRead), 0, (buffSize - bytesRead));


    hipMemcpy(buffGPU, buffer, sizeof(unsigned char) * buffSize, hipMemcpyHostToDevice);
    hipMemcpy(keysGPU, exp_key, sizeof(unsigned char) * EXP_KEY_SIZE, hipMemcpyHostToDevice);

    if (buffSize > CACHE_SIZE) {
      int nBlkCache = (buffSize / CACHE_SIZE) + 1;
      for(int i = 0; i < nBlkCache; i++) {
        /* Algoritmo aqui */
        int nBlocks = 3;
        int nTh = 1024;

        aes<<<nBlocks, nTh>>>(buffGPU + CACHE_SIZE * i, keysGPU, 3072);
        hipDeviceSynchronize();
        hipMemcpy(buffer + CACHE_SIZE * i, buffGPU + CACHE_SIZE * i, sizeof(unsigned char) * CACHE_SIZE, hipMemcpyDeviceToHost);
      }
    }
    else {
      /* Algoritmo aqui */
      int nBlocks = 1;
      int nTh = buffSize / 16;

      if (nTh > MAX_THR_PBLK) {
        nBlocks = (nTh / MAX_THR_PBLK) + 1;
        nTh = MAX_THR_PBLK;
      }

      aes<<<nBlocks, nTh>>>(buffGPU, keysGPU, buffSize / 16);

      hipMemcpy(buffer, buffGPU, sizeof(unsigned char) * buffSize, hipMemcpyDeviceToHost);
    }

    //DEBUG
    for(int i = 0; i < buffSize; i += 16)
      printState(buffer + i, 16);

    memset(buffer, 0, buffSize * sizeof(unsigned char));
    bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
  }

  //Libera a memória alocada
  hipFree(buffGPU);
  hipFree(keysGPU);
  delete [] buffer;
  fclose(fin);
  return 0;
}
