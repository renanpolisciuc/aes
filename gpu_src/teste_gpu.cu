#include<stdio.h>
#include<iostream>
#include<string.h>
#include "aes_gpu.h"

using namespace std;

//#define MAX_BUFFER_SIZE  536870912
#define MAX_BUFFER_SIZE  CACHE_SIZE
#define MAX_THR_PBLK 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__host__
void printState(unsigned char * state, int size) {
  for(int i = 0; i < size; i++)
    printf("%X ", state[i]);
  printf("\n");
}

int main(int argc, char ** argv) {
  long buffSize = 0L, //Tamanho do arquivo
       bytesRead = 0L; //Quantidade de bytes lidos pelo fread

  FILE * fin; // Pointer para o arquivo
  FILE * fout;
  unsigned char * buffer = NULL; //Bytes do arquivo
  unsigned char * buffGPU = NULL;
  unsigned char * keysGPU = NULL;

  unsigned char key[16] = {
    1, 2, 3, 4,
    5, 6, 7, 8,
    9, 10, 11, 12,
    13, 14, 15, 16
  };

  hipEvent_t start, stop;
  float time_run = 0.0, time_total = 0.0;

  fin = fopen(argv[1], "r"); //Abre o arquivo como leitura
  fout = fopen("gpu.out", "w"); //Abre o arquivo como escrita

  //Verificar se o arquivo foi 'aberto'
  if (fin == NULL) {
    cout << "Ocorreu uma falha ao tentar abrir o arquivo " << argv[1] << endl;
    return -1;
  }

  //Expansão de chaves
  unsigned char exp_key[EXP_KEY_SIZE];
  addKeyExpansion(key, exp_key);

  //Lê os bytes do arquivo e adiciona padding caso necessário
  buffer = new unsigned char[MAX_BUFFER_SIZE];
  bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
  gpuErrchk(hipMalloc((void**)&buffGPU, sizeof(unsigned char) * MAX_BUFFER_SIZE));
  gpuErrchk(hipMalloc((void**)&keysGPU, sizeof(unsigned char) * EXP_KEY_SIZE));
  gpuErrchk(hipMemcpy(keysGPU, exp_key, sizeof(unsigned char) * EXP_KEY_SIZE, hipMemcpyHostToDevice));
  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));

  while (bytesRead > 0) {
    if (bytesRead < MAX_BUFFER_SIZE)
      bytesRead -= 1; /* Se ler o último bloco do arquivo, desconsidera o EOF */

    buffSize = bytesRead;

    //Verifica se o tamanho do arquivo é múltiplo de 16. Caso não for, encontra o próximo múltiplo de 16 a partir
    //do tamanho do arquivo
    if (buffSize % 16 != 0)
      buffSize = buffSize + (16 - (buffSize % 16));

    if (bytesRead < buffSize)
      memset((buffer + bytesRead), 0, (buffSize - bytesRead));

    gpuErrchk(hipMemcpy(buffGPU, buffer, sizeof(unsigned char) * buffSize, hipMemcpyHostToDevice));
    if (buffSize > CACHE_SIZE) {
      int nBlkCache = (buffSize / CACHE_SIZE) + 1;
      for(int i = 0; i < nBlkCache; i++) {
        /* Algoritmo aqui */
        int nBlocks = 3;
        int nTh = 1024;
        gpuErrchk(hipEventRecord(start));
        aes<<<nBlocks, nTh>>>(buffGPU + CACHE_SIZE * i, keysGPU, 3072);
        gpuErrchk(hipEventRecord(stop));
        gpuErrchk(hipEventSynchronize(stop));
        gpuErrchk(hipEventElapsedTime(&time_run, start, stop));
        time_total += time_run;
        gpuErrchk(hipMemcpy(buffer + CACHE_SIZE * i, buffGPU + CACHE_SIZE * i, sizeof(unsigned char) * CACHE_SIZE, hipMemcpyDeviceToHost));
      }
    }
    else {
      /* Algoritmo aqui */
      int nBlocks = 1;
      int nTh = buffSize / 16;

      if (nTh > MAX_THR_PBLK) {
        nBlocks = (nTh / MAX_THR_PBLK) + 1;
        nTh = MAX_THR_PBLK;
      }
      gpuErrchk(hipEventRecord(start));
      aes<<<nBlocks, nTh>>>(buffGPU, keysGPU, buffSize / 16);
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&time_run, start, stop));
      time_total += time_run;
      gpuErrchk(hipMemcpy(buffer, buffGPU, sizeof(unsigned char) * buffSize, hipMemcpyDeviceToHost));
    }
    // for(int i = 0; i < buffSize; i += 16)
    //   printState(buffer + i, 16);
    fwrite(buffer, sizeof(unsigned char), buffSize, fout);
    memset(buffer, 0, MAX_BUFFER_SIZE * sizeof(unsigned char));
    bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
  }
  cout << time_total << endl;
  //Libera a memória alocada
  hipFree(buffGPU);
  hipFree(keysGPU);
  delete [] buffer;
  fclose(fin);
  fclose(fout);
  return 0;
}
