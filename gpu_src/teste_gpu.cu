#include<stdio.h>
#include<iostream>
#include<string.h>
#include <chrono>
#include "aes_gpu.h"

using namespace std;
using namespace std::chrono;

#define MAX_BUFFER_SIZE  536870912
//#define MAX_BUFFER_SIZE  CACHE_SIZE
#define MAX_THR_PBLK 1024

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__host__
void printState(unsigned char * state, int size) {
  for(int i = 0; i < size; i++)
    printf("%X ", state[i]);
  printf("\n");
}

__host__
int getProximoMultiplo16(long numero) {
  int n = numero;
  if (n % 16 != 0)
    n = n + (16 - (n % 16));
  return n;
}

int main(int argc, char ** argv) {
  long MAX_THREADS = MAX_THR_PBLK;
  long buffSize = 0L, //Tamanho do arquivo
       bytesRead = 0L; //Quantidade de bytes lidos pelo fread
  long long total_bytes = 0L;
  double duracao_IO = 0L;

  FILE * fin; // Pointer para o arquivo
  FILE * fout;
  unsigned char * buffer = NULL; //Bytes do arquivo
  unsigned char * buffGPU = NULL;
  unsigned char * keysGPU = NULL;

  unsigned char key[16] = {
    1, 2, 3, 4,
    5, 6, 7, 8,
    9, 10, 11, 12,
    13, 14, 15, 16
  };

  hipEvent_t start, stop, start_all, stop_all;
  float time_run = 0.0, time_total = 0.0, time_total_processamento = 0.0;

  fin = fopen(argv[1], "r"); //Abre o arquivo como leitura
  fout = fopen("gpu.out", "w"); //Abre o arquivo como escrita

  //Verificar se o arquivo foi 'aberto'
  if (fin == NULL) {
    cout << "Ocorreu uma falha ao tentar abrir o arquivo " << argv[1] << endl;
    return -1;
  }

  int countDevices = 0;
  hipGetDeviceCount(&countDevices);
  for(int i = 0; i < countDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);

    MAX_THREADS = prop.maxThreadsPerBlock;
    break;
  }
  //Expansão de chaves
  unsigned char * exp_key(NULL);
  HANDLE_ERROR(hipHostMalloc(&exp_key, sizeof(unsigned char) * EXP_KEY_SIZE));

  addKeyExpansion(key, exp_key);

  //Lê os bytes do arquivo e adiciona padding caso necessário
  HANDLE_ERROR(hipHostMalloc(&buffer, MAX_BUFFER_SIZE * sizeof(unsigned char)));
  bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
  HANDLE_ERROR(hipMalloc((void**)&buffGPU, sizeof(unsigned char) * MAX_BUFFER_SIZE));
  HANDLE_ERROR(hipMalloc((void**)&keysGPU, sizeof(unsigned char) * EXP_KEY_SIZE));
  HANDLE_ERROR(hipMemcpy(keysGPU, exp_key, sizeof(unsigned char) * EXP_KEY_SIZE, hipMemcpyHostToDevice));

  HANDLE_ERROR(hipEventCreate(&start_all));
  HANDLE_ERROR(hipEventCreate(&stop_all));
  HANDLE_ERROR(hipEventRecord(start_all, 0));

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  while (bytesRead > 0) {

    total_bytes += bytesRead;
    if (bytesRead < MAX_BUFFER_SIZE)
      bytesRead -= 1; /* Se ler o último bloco do arquivo, desconsidera o EOF */

    buffSize = bytesRead;

    //Verifica se o tamanho do arquivo é múltiplo de 16. Caso não for, encontra o próximo múltiplo de 16 a partir
    //do tamanho do arquivo
    buffSize = getProximoMultiplo16(buffSize);

    auto t11_io = high_resolution_clock::now();
    if (bytesRead < buffSize)
      memset((buffer + bytesRead), 0, (buffSize - bytesRead));

    HANDLE_ERROR(hipMemcpy(buffGPU, buffer, sizeof(unsigned char) * buffSize, hipMemcpyHostToDevice));
    auto t21_io = high_resolution_clock::now();
    auto duration_IO1 = duration_cast<milliseconds>( t21_io - t11_io );
    duracao_IO += duration_IO1.count();

    int nBlocks = 1;
    int nThreads = buffSize / 16;

    if (nThreads > MAX_THREADS) {
      nBlocks = (nThreads / MAX_THREADS) + 1;
      nThreads = MAX_THREADS;
    }

    aes<<<nBlocks, nThreads>>>(buffGPU, keysGPU, buffSize / 16);

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time_run, start, stop));

    time_total_processamento += time_run;

    // for(int i = 0; i < buffSize; i += 16)
    //   printState(buffer + i, 16);
    auto t12_io = high_resolution_clock::now();
    HANDLE_ERROR(hipMemcpy(buffer, buffGPU, sizeof(unsigned char) * buffSize, hipMemcpyDeviceToHost));
    fwrite(buffer, sizeof(unsigned char), buffSize, fout);
    memset(buffer, 0, MAX_BUFFER_SIZE * sizeof(unsigned char));
    bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
    auto t22_io = high_resolution_clock::now();
    auto duration_IO2 = duration_cast<milliseconds>( t22_io - t12_io );
    duracao_IO += duration_IO2.count();
  }

  HANDLE_ERROR(hipEventRecord(stop_all, 0));
  HANDLE_ERROR(hipEventSynchronize(stop_all));
  HANDLE_ERROR(hipEventElapsedTime(&time_total, start_all, stop_all));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  HANDLE_ERROR(hipEventDestroy(start_all));
  HANDLE_ERROR(hipEventDestroy(stop_all));

  cout <<  ((double)total_bytes / 1000000000)   << " | " << (float) (time_total / 1000) << " | " << (float)(time_total_processamento / 1000)<< " | " << (float) (duracao_IO / 1000) << endl;

  //Libera a memória alocada
  hipFree(buffGPU);
  hipFree(keysGPU);
  hipHostFree(exp_key);
  hipHostFree(buffer);
  fclose(fin);
  fclose(fout);
  return 0;
}
