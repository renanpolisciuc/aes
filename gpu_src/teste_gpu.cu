#include<stdio.h>
#include<iostream>
#include<string.h>
#include <chrono>
#include "aes_gpu.h"

using namespace std;
using namespace std::chrono;

#define MAX_BUFFER_SIZE  536870912
//#define MAX_BUFFER_SIZE  CACHE_SIZE
#define MAX_THR_PBLK 1024

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__host__
void printState(unsigned char * state, int size) {
  for(int i = 0; i < size; i++)
    printf("%X ", state[i]);
  printf("\n");
}

int main(int argc, char ** argv) {
  long buffSize = 0L, //Tamanho do arquivo
       bytesRead = 0L; //Quantidade de bytes lidos pelo fread
  long long total_bytes = 0L;

  FILE * fin; // Pointer para o arquivo
  FILE * fout;
  unsigned char * buffer = NULL; //Bytes do arquivo
  unsigned char * buffGPU = NULL;
  unsigned char * keysGPU = NULL;
  float duracao = 0.0;

  unsigned char key[16] = {
    1, 2, 3, 4,
    5, 6, 7, 8,
    9, 10, 11, 12,
    13, 14, 15, 16
  };

  hipEvent_t start, stop;
  float time_run = 0.0, time_total = 0.0;

  fin = fopen(argv[1], "r"); //Abre o arquivo como leitura
  fout = fopen("gpu.out", "w"); //Abre o arquivo como escrita

  //Verificar se o arquivo foi 'aberto'
  if (fin == NULL) {
    cout << "Ocorreu uma falha ao tentar abrir o arquivo " << argv[1] << endl;
    return -1;
  }

  //Expansão de chaves
  unsigned char exp_key[EXP_KEY_SIZE];
  addKeyExpansion(key, exp_key);

  //Lê os bytes do arquivo e adiciona padding caso necessário
  buffer = new unsigned char[MAX_BUFFER_SIZE];
  bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
  HANDLE_ERROR(hipMalloc((void**)&buffGPU, sizeof(unsigned char) * MAX_BUFFER_SIZE));
  HANDLE_ERROR(hipMalloc((void**)&keysGPU, sizeof(unsigned char) * EXP_KEY_SIZE));
  HANDLE_ERROR(hipMemcpy(keysGPU, exp_key, sizeof(unsigned char) * EXP_KEY_SIZE, hipMemcpyHostToDevice));


  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));
  while (bytesRead > 0) {

    total_bytes += bytesRead;
    if (bytesRead < MAX_BUFFER_SIZE)
      bytesRead -= 1; /* Se ler o último bloco do arquivo, desconsidera o EOF */

    buffSize = bytesRead;

    //Verifica se o tamanho do arquivo é múltiplo de 16. Caso não for, encontra o próximo múltiplo de 16 a partir
    //do tamanho do arquivo
    if (buffSize % 16 != 0)
      buffSize = buffSize + (16 - (buffSize % 16));

    if (bytesRead < buffSize)
      memset((buffer + bytesRead), 0, (buffSize - bytesRead));

    HANDLE_ERROR(hipMemcpy(buffGPU, buffer, sizeof(unsigned char) * buffSize, hipMemcpyHostToDevice));

    if (buffSize >= CACHE_SIZE) {
      int nBlkCache = (buffSize / CACHE_SIZE);
      for(int i = 0; i < nBlkCache; i++) {
        /* Algoritmo aqui */
        int nBlocks = 3;
        int nTh = 1024;

        aes<<<nBlocks, nTh>>>(buffGPU + CACHE_SIZE * i, keysGPU, 3072);
        HANDLE_ERROR(hipMemcpy(buffer + CACHE_SIZE * i, buffGPU + CACHE_SIZE * i, sizeof(unsigned char) * CACHE_SIZE, hipMemcpyDeviceToHost));
      }
    }
    else {
      /* Algoritmo aqui */
      int nBlocks = 1;
      int nTh = buffSize / 16;

      if (nTh > MAX_THR_PBLK) {
        nBlocks = (nTh / MAX_THR_PBLK) + 1;
        nTh = MAX_THR_PBLK;
      }

      aes<<<nBlocks, nTh>>>(buffGPU, keysGPU, buffSize / 16);
      time_total += time_run;
      HANDLE_ERROR(hipMemcpy(buffer, buffGPU, sizeof(unsigned char) * buffSize, hipMemcpyDeviceToHost));
    }
    fwrite(buffer, sizeof(unsigned char), buffSize, fout);
    memset(buffer, 0, MAX_BUFFER_SIZE * sizeof(unsigned char));
    bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
  }

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&time_run, start, stop));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  cout <<  ((double)total_bytes / 1000000000)   << " | " << (float) (time_run / 1000) << endl;

  //Libera a memória alocada
  hipFree(buffGPU);
  hipFree(keysGPU);
  delete [] buffer;
  fclose(fin);
  fclose(fout);
  return 0;
}
