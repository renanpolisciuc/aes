#include<stdio.h>
#include<iostream>
#include<string.h>
#include "aes_gpu.h"

using namespace std;

#define MAX_BUFFER_SIZE  536870912
__host__
void printState(unsigned char * state, int size) {
  for(int i = 0; i < size; i++)
    printf("%X ", state[i]);
  printf("\n");
}

int main(int argc, char ** argv) {
  long buffSize = 0L, //Tamanho do arquivo
       bytesRead = 0L,
       fileSize = 0L; //Quantidade de bytes lidos pelo fread

  FILE * fin; // Pointer para o arquivo
  unsigned char * buffer = NULL; //Bytes do arquivo
  unsigned char * buffGPU = NULL, *buffGPUOut = NULL;
  unsigned char * keysGPU = NULL;

  unsigned char key[16] = {
    1, 2, 3, 4,
    5, 6, 7, 8,
    9, 10, 11, 12,
    13, 14, 15, 16
  };

  fin = fopen(argv[1], "r"); //Abre o arquivo como leitura

  //Verificar se o arquivo foi 'aberto'
  if (fin == NULL) {
    cout << "Ocorreu uma falha ao tentar abrir o arquivo " << argv[1] << endl;
    return -1;
  }

  /**
    Descobre o tamanho do arquivo para debug
  */
  fseek(fin, 0L, SEEK_END);
  fileSize = ftell(fin) - 1;
  rewind(fin);

  //Expansão de chaves
  unsigned char exp_key[EXP_KEY_SIZE];
  addKeyExpansion(key, exp_key);

  //Lê os bytes do arquivo e adiciona padding caso necessário
  buffer = new unsigned char[MAX_BUFFER_SIZE];
  bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin) - 1;
  buffSize = bytesRead;
  if (bytesRead % 16 != 0)
    buffSize = bytesRead + (16 - (bytesRead % 16));

  if (bytesRead < buffSize)
    memset((buffer + bytesRead), 0, (buffSize - bytesRead));

  hipMalloc((void**)&buffGPU, sizeof(unsigned char) * buffSize);
  hipMalloc((void**)&buffGPUOut, sizeof(unsigned char) * buffSize);
  hipMalloc((void**)&keysGPU, sizeof(unsigned char) * EXP_KEY_SIZE);
  hipMemcpy(buffGPU, buffer, sizeof(unsigned char) * buffSize, hipMemcpyHostToDevice);
  hipMemcpy(keysGPU, exp_key, sizeof(unsigned char) * EXP_KEY_SIZE, hipMemcpyHostToDevice);
  /* Algoritmo aqui */
  int nBlocks = 1;
  int nTh = buffSize / 16;

  aes<<<nBlocks, nTh>>>(buffGPU, buffGPUOut, keysGPU, nTh);

  hipMemcpy(buffer, buffGPUOut, sizeof(unsigned char) * buffSize, hipMemcpyDeviceToHost);

  //DEBUG
  for(int i = 0; i < buffSize; i += 16)
    printState(buffer + i, 16);

  //Libera a memória alocada
  hipFree(buffGPU);
  hipFree(buffGPUOut);
  hipFree(keysGPU);
  delete [] buffer;
  fclose(fin);
  return 0;
}
