#include<stdio.h>
#include<iostream>
#include<string.h>
#include <chrono>
#include "aes_gpu.h"

using namespace std;
using namespace std::chrono;

#define MAX_BUFFER_SIZE  getProximoMultiplo16(536870912)
//#define MAX_BUFFER_SIZE  CACHE_SIZE
#define MAX_THR_PBLK 1024

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__host__
void printState(unsigned char * state, int size) {
  for(int i = 0; i < size; i++)
    printf("%X ", state[i]);
  printf("\n");
}

__host__
int getProximoMultiplo16(long numero) {
  int n = numero;
  if (n % 16 != 0)
    n = n + (16 - (n % 16));
  return n;
}

int main(int argc, char ** argv) {
  long buffSize = 0L, //Tamanho do arquivo
       bytesRead = 0L; //Quantidade de bytes lidos pelo fread
  long long total_bytes = 0L;

  FILE * fin; // Pointer para o arquivo
  FILE * fout;
  unsigned char * buffer = NULL; //Bytes do arquivo
  unsigned char * buffGPU = NULL;
  unsigned char * keysGPU = NULL;

  unsigned char key[16] = {
    1, 2, 3, 4,
    5, 6, 7, 8,
    9, 10, 11, 12,
    13, 14, 15, 16
  };

  hipEvent_t start, stop;
  float time_run = 0.0, time_total = 0.0;

  fin = fopen(argv[1], "r"); //Abre o arquivo como leitura
  fout = fopen("gpu.out", "w"); //Abre o arquivo como escrita

  //Verificar se o arquivo foi 'aberto'
  if (fin == NULL) {
    cout << "Ocorreu uma falha ao tentar abrir o arquivo " << argv[1] << endl;
    return -1;
  }

  //Expansão de chaves
  unsigned char exp_key[EXP_KEY_SIZE];
  addKeyExpansion(key, exp_key);

  //Lê os bytes do arquivo e adiciona padding caso necessário
  buffer = new unsigned char[MAX_BUFFER_SIZE];
  bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);
  HANDLE_ERROR(hipMalloc((void**)&buffGPU, sizeof(unsigned char) * MAX_BUFFER_SIZE));
  HANDLE_ERROR(hipMalloc((void**)&keysGPU, sizeof(unsigned char) * EXP_KEY_SIZE));
  HANDLE_ERROR(hipMemcpy(keysGPU, exp_key, sizeof(unsigned char) * EXP_KEY_SIZE, hipMemcpyHostToDevice));


  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  while (bytesRead > 0) {

    total_bytes += bytesRead;
    if (bytesRead < MAX_BUFFER_SIZE)
      bytesRead -= 1; /* Se ler o último bloco do arquivo, desconsidera o EOF */

    buffSize = bytesRead;

    //Verifica se o tamanho do arquivo é múltiplo de 16. Caso não for, encontra o próximo múltiplo de 16 a partir
    //do tamanho do arquivo
    buffSize = getProximoMultiplo16(buffSize);

    if (bytesRead < buffSize)
      memset((buffer + bytesRead), 0, (buffSize - bytesRead));

    HANDLE_ERROR(hipMemcpy(buffGPU, buffer, sizeof(unsigned char) * MAX_BUFFER_SIZE, hipMemcpyHostToDevice));

    int nBlocks = 1;
    int nTh = buffSize / 16;

    if (nTh > MAX_THR_PBLK) {
      nBlocks = (nTh / MAX_THR_PBLK) + 1;
      nTh = MAX_THR_PBLK;
    }
    aes<<<nBlocks, nTh>>>(buffGPU, keysGPU, (buffSize / 16));
    time_total += time_run;
    HANDLE_ERROR(hipMemcpy(buffer, buffGPU, sizeof(unsigned char) * MAX_BUFFER_SIZE, hipMemcpyDeviceToHost));

    // for(int i = 0; i < buffSize; i += 16)
    //   printState(buffer + i, 16);
    fwrite(buffer, sizeof(unsigned char), buffSize, fout);
    memset(buffer, 0, MAX_BUFFER_SIZE * sizeof(unsigned char));
    bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin);

  }

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&time_run, start, stop));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  cout <<  ((double)total_bytes / 1000000000)   << " | " << (float) (time_run / 1000) << endl;

  //Libera a memória alocada
  hipFree(buffGPU);
  hipFree(keysGPU);
  delete [] buffer;
  fclose(fin);
  fclose(fout);
  return 0;
}
