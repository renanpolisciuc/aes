#include<stdio.h>
#include<iostream>
#include<string.h>
#include "aes_gpu.h"
#include "key_expansion.h"
using namespace std;

#define MAX_BUFFER_SIZE  536870912

//DEBUG
void printState(unsigned char * state, int size);

void printState(unsigned char * state, int size) {
  for(int i = 0; i < size; i++)
    printf("%X ", state[i]);
  cout << endl;
}

int main(int argc, char ** argv) {
  long buffSize = 0L, //Tamanho do arquivo
       bytesRead = 0L,
       fileSize = 0L; //Quantidade de bytes lidos pelo fread

  FILE * fin; // Pointer para o arquivo
  unsigned char * buffer = NULL; //Bytes do arquivo
  unsigned char * buffGPU = NULL;

  unsigned char key[16] = {
    1, 2, 3, 4,
    5, 6, 7, 8,
    9, 10, 11, 12,
    13, 14, 15, 16
  };

  fin = fopen(argv[1], "r"); //Abre o arquivo como leitura

  //Verificar se o arquivo foi 'aberto'
  if (fin == NULL) {
    cout << "Ocorreu uma falha ao tentar abrir o arquivo " << argv[1] << endl;
    return -1;
  }

  /**
    Descobre o tamanho do arquivo para debug
  */
  fseek(fin, 0L, SEEK_END);
  fileSize = ftell(fin) - 1;
  rewind(fin);

  //Expansão de chaves
  unsigned char exp_key[EXP_KEY_SIZE];
  addKeyExpansion(key, exp_key);

  //Lê os bytes do arquivo e adiciona padding caso necessário
  buffer = new unsigned char[MAX_BUFFER_SIZE];
  bytesRead = fread(buffer, sizeof(unsigned char), MAX_BUFFER_SIZE, fin) - 1;
  buffSize = bytesRead;
  if (bytesRead % 16 != 0)
    buffSize = bytesRead + (16 - (bytesRead % 16));

  if (bytesRead < buffSize)
    memset((buffer + bytesRead), 'c', (buffSize - bytesRead));

  hipMalloc((void**)&buffGPU, sizeof(unsigned char) * buffSize);
  hipMemcpy(buffGPU, buffer, sizeof(unsigned char) * buffSize, hipMemcpyHostToDevice);
  /* Algoritmo aqui */
  int nBlocks = 1;
  int nTh = buffSize / 16;

  aes<<<nBlocks, nTh>>>(buffGPU, exp_key, nTh);

  hipMemcpy(buffer, buffGPU, sizeof(unsigned char) * buffSize, hipMemcpyDeviceToHost);
  cout << buffer << endl;
  //Libera a memória alocada
  hipFree(buffGPU);
  delete [] buffer;
  fclose(fin);
  return 0;
}
