#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include "aes_gpu.h"
#include "tables_gpu.h"

__device__
void subBytes(unsigned char * state) {
  //Faz a substituição byte a byte pela S_BOX
  for(int i = 0; i < 16; i++)
    state[i] = S_BOX[state[i]];
}

__device__
void shiftRows(unsigned char * state) {
}

__device__
void addRoundKey(unsigned char * state, unsigned char * key) {
  //Xor byte a byte entre o estado e a chave
  for(int i = 0; i < 16; i++)
    state[i] ^= key[i];
}

__device__
void rotWord(unsigned char * word) {
  unsigned char tmp_word = word[0];
  //Rotaciona uma word
  word[0] = word[1];
  word[1] = word[2];
  word[2] = word[3];
  word[3] = tmp_word;
}

__device__
void subWord(unsigned char * word) {
  //Substitui cada byte da word por um byte da S_BOX
  for(int i = 0; i < 4; i++)
    word[i] = S_BOX[word[i]];
}

__device__
void addKeyExpansionCore(unsigned char * key, unsigned char i) {
  //Rotaciona, substitui e faz um xor com a tabela rcon (apenas os bits mais à esquerda)
  rotWord(key);
  subWord(key);
  key[0] ^= rcon[i];
}

__device__
void translateWord(unsigned char * word) {

}

__device__
void addKeyExpansion(unsigned char * key, unsigned char * exp_keys) {

}

__device__
void mixColumns(unsigned char * state) {
  //Algoritmo mix column
  //Operação em GF(2^8)
}

__global__
void aes(unsigned char * in_bytes, unsigned char * key) {

}
