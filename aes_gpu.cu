#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include "aes_gpu.h"
#include "tables_gpu.h"

__device__
void subBytes(unsigned char * state) {
  //Faz a substituição byte a byte pela S_BOX
  for(int i = 0; i < 16; i++)
    state[i] = S_BOX[state[i]];
}

__device__
void shiftRows(unsigned char * state) {
  unsigned char tmp[16];
  //0 shifts
  tmp[0] = state[0];
  tmp[4] = state[4];
  tmp[8] = state[8];
  tmp[12] = state[12];

  //1 shift para a esquerda
  tmp[1] = state[5];
  tmp[5] = state[9];
  tmp[9] = state[13];
  tmp[13] = state[1];

  //2 shifts para a esquerda
  tmp[2] = state[10];
  tmp[6] = state[14];
  tmp[10] = state[2];
  tmp[14] = state[6];

  //3 shifts para a esquerda
  tmp[3] = state[15];
  tmp[7] = state[3];
  tmp[11] = state[7];
  tmp[15] = state[11];
  hipMemcpy(state, tmp, 16, hipMemcpyDeviceToDevice);
}

__device__
void addRoundKey(unsigned char * state, unsigned char * key) {
  //Xor byte a byte entre o estado e a chave
  for(int i = 0; i < 16; i++)
    state[i] ^= key[i];
}
__device__
void mixColumns(unsigned char * state) {
  //Algoritmo mix column
  //Operação em GF(2^8)
}

__global__
void aes(unsigned char * in_bytes, unsigned char * key) {

}
